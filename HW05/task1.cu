#include <cstdio>
#include <cstdlib>
#include <ctime>
#include "reduce.cuh"

#define MAX_RAND 1000

int main(int argc, char* argv[]) {

    std::srand((unsigned int)std::time(NULL));
    
    std::size_t N = std::atoll(argv[1]);
    std::size_t threads_per_block = std::atoll(argv[2]);    

    std::size_t num_blocks = (N+(2*threads_per_block-1))/(2*threads_per_block);

    //Allocating memory to host arrays
    float * input = (float *) std::malloc(N*sizeof(float));
    float * output = (float *) std::malloc(num_blocks*sizeof(float));
    
    //Populating values within the host arrays
    for(std::size_t i=0;i<N;i++) {
        input[i] = ((float(std::rand()%MAX_RAND)*2.0)/MAX_RAND)-1.0; //Value between -1.0 and 1.0
    }  

    // Declare pointers that will point to the memory allocated on the device.
    float *dInput,*dOutput;

 
    // Allocate memory on the device for input
    hipMalloc(&dInput, sizeof(float)*N);

    //Final result array on device
    hipMalloc(&dOutput, sizeof(float)*num_blocks);

    //Copy data from host to device
    hipMemcpy(dInput, input, sizeof(float)*N, hipMemcpyHostToDevice);

    //For recording time:
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start); //START EVENT

    // Launch the kernel on the device
    
    reduce(&dInput, &dOutput, N, threads_per_block);

    hipEventRecord(stop); //STOP EVENT
    hipEventSynchronize(stop);

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    // Copy the output array back from the device to the host and print its values
    hipMemcpy(output, dOutput, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(input, dInput, sizeof(float), hipMemcpyDeviceToHost);

    //Print results
    std::printf("%f\n", input[0]);
    std::printf("%f\n", ms); 
 
    // Free resources
    hipFree(dInput);
    hipFree(dOutput);
    std::free(input);
    std::free(output);

    return 0;
}
