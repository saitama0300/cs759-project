
#include <hip/hip_runtime.h>
#include<cstdio>

__global__ void reduce_kernel(float *g_idata, float *g_odata, unsigned int n) {

    extern __shared__ float shared_memory[];
    float *s_idata = shared_memory;

    //For add on Load (Reference #4)
    std::size_t i = blockDim.x*(2*blockIdx.x) + threadIdx.x;

    //Populate the shared memory
    if(i<n) {
        if(i+blockDim.x<n){
            //Add on Load (Reference #4)
            s_idata[threadIdx.x] = g_idata[i]+g_idata[i+blockDim.x];
        }
        else {
            //Corner Case
            s_idata[threadIdx.x] = g_idata[i];
        }
    }
    else {
        //Corner Case
        s_idata[threadIdx.x] = 0.0;
    }

    __syncthreads();
  
    //Calculate partial sum across s_idata
    for(std::size_t j=blockDim.x/2;j>0;j>>=1) {
        if(threadIdx.x<j){
            s_idata[threadIdx.x] += s_idata[threadIdx.x+j];
        }
     
        __syncthreads();
    }
 
    //Populate g_odata & g_idata with new final sum (stored in s_idata
    g_odata[blockIdx.x] = s_idata[0];
    g_idata[blockIdx.x] = s_idata[0];

    __syncthreads();
}
__host__ void reduce(float **input, float **output, unsigned int N,
                     unsigned int threads_per_block){

    //Initializing loop variables
    std::size_t num_blocks = (N+(2*threads_per_block-1))/(2*threads_per_block); 
    std::size_t input_size = N;

    while(input_size>1){

        //kernel launch
        reduce_kernel<<<num_blocks,threads_per_block,threads_per_block*sizeof(float)>>>(*(input), *(output), input_size);
        hipDeviceSynchronize();
        //Input size change for next kernel launch
        input_size = num_blocks;
        //Get new number of blocks every iteration
        num_blocks = (input_size+(2*threads_per_block-1))/(2*threads_per_block); 
    }

}