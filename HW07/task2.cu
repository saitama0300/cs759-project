#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "count.cuh"

#define MAX_RAND 501

int main(int argc, char* argv[]) {

    std::srand((unsigned int)std::time(NULL));

    //Fetch n from the argument
    std::size_t n = std::atoll(argv[1]);

    //Create thrust host vector here
    thrust::host_vector<int> h_vector(n);

    //Initialize the h_vector
    for(std::size_t i=0;i<n;i++) {
        h_vector[i] = (int(std::rand()%MAX_RAND)); //RANDOM VALUE
    }  

    //Instantiate device vector, value vector & count storing vector
    thrust::device_vector<int> d_vector = h_vector;
    thrust::device_vector<int> values(n);
    thrust::device_vector<int> counts(n);
    
    //Needed for value to be used in the count function
    values = d_vector;

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start); //START EVENT

    // Call to the function
    count(d_vector,values,counts);

    hipEventRecord(stop); //STOP EVENT
    hipEventSynchronize(stop);

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    //Print final results
    std::cout << values.back() << std::endl;
    std::cout << counts.back() << std::endl; 
    std::printf("%f\n", ms); 
}