//Large Part of the code has been copied from Prof. Dan's repository

#define HIPCUB_STDERR // print CUDA runtime errors to console
#include <stdio.h>
#include <cub/util_allocator.cuh>
#include <cub/device/device_reduce.cuh>
#include "cub/util_debug.cuh"

#define MAX_RAND 1000

using namespace hipcub;
CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory

int main(int argc, char* argv[]) {
    std::srand((unsigned int)std::time(NULL));

    //Fetch n from the argument
    std::size_t n = std::atoll(argv[1]);

    //Host array pointer
    float * h_in = (float *) std::malloc(n*sizeof(float));

    //Initialize the h_vector
    for(std::size_t i=0;i<n;i++) {
        h_in[i] = ((float(std::rand()%MAX_RAND)*2.0)/MAX_RAND)-1.0; //RANDOM VALUE
    }  

    // Set up device arrays
    float* d_in = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)& d_in, sizeof(float) * n));

    // Initialize device input
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(float) * n, hipMemcpyHostToDevice));

    // Setup device output array
    float* d_sum = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)& d_sum, sizeof(float) * 1));
    // Request and allocate temporary storage
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    CubDebugExit(DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_sum, n));
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

    //Recording Time
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start); //START EVENT
    // Do the actual reduce operation

    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_sum, n);

    hipEventRecord(stop); //STOP EVENT
    hipEventSynchronize(stop);
    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    float result;
    CubDebugExit(hipMemcpy(&result, d_sum, sizeof(float) * 1, hipMemcpyDeviceToHost));
    // Check for correctness
    printf("%f\n", result);
    printf("%f\n", ms);

    // Cleanup
    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
    if (d_sum) CubDebugExit(g_allocator.DeviceFree(d_sum));
    if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));
    
    return 0;
}
