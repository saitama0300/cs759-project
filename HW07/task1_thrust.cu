#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#define MAX_RAND 1000

int main(int argc, char* argv[]) {

    std::srand((unsigned int)std::time(NULL));

    //Fetch n from the argument
    std::size_t n = std::atoll(argv[1]);

    //Create thrust host vector here-
    thrust::host_vector<float> h_vector(n);

    //Initialize the h_vector
    for(std::size_t i=0;i<n;i++) {
        h_vector[i] = ((float(std::rand()%MAX_RAND)*2.0)/MAX_RAND)-1.0; //RANDOM VALUE
    }  

    //Instantiate device vector
    thrust::device_vector<float> d_vector = h_vector;
    
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start); //START EVENT

    // Call to the function for reduction (you add here so use plus functor)
    float result = thrust::reduce(d_vector.begin() , d_vector.end(), 0.0, thrust::plus<float>());

    hipEventRecord(stop); //STOP EVENT
    hipEventSynchronize(stop);

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    //Print final results
    std::printf("%f\n", result);  
    std::printf("%f\n", ms); 
}