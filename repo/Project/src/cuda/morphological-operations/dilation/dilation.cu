#include <iostream>
#include <string>

#include "dilationfilter.cuh"

#define STB_IMAGE_IMPLEMENTATION
#include "../../../image.hpp"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../../../image_write.hpp"

using namespace std;

int main(int argc, char *argv[]) {

            char* inputImg = argv[1];
            char* outputImg = argv[2];  
            std::size_t threads_per_block = atoll(argv[3]);   
            
            int width, height, channel;

            //WHY DO WE USE UNSIGNED CHAR ? AS EACH RGB PIXEL VALUE IS BETWEEN 0 and 255. HENCE USING LARGER DATATYPES MAKES
            //NO SENSE UNNECCESARILY TO STORE THE IMAGE IN GLOBAL/MAIN MEMORY AS IMAGE FILES ARE ANYWAYS LARGE IN SIZE

            cout<<"-- DILATION-- 3x3"<<endl;
            unsigned char* rgb_image = stbi_load(inputImg, &width, &height, &channel, 3); // 3 means RGB

            unsigned char* input_img_device;
            unsigned char* output_img;

            hipMalloc(&input_img_device, sizeof(unsigned char)*height*width*NUM_CHANNELS);
            hipMallocManaged(&output_img, sizeof(unsigned char)*height*width*NUM_CHANNELS);

            hipMemcpy(input_img_device, rgb_image, sizeof(unsigned char)*height*width*NUM_CHANNELS, hipMemcpyHostToDevice);

            cout<<"IMAGE FILE: "<<inputImg<<endl;
            cout<<"WIDTH OF IMAGE: "<<width<<endl;
            cout<<"HEIGHT OF IMAGE: "<<height<<endl;

            dilationfilter(&input_img_device, &output_img, height, width, threads_per_block);

            stbi_write_jpg(outputImg, width, height, channel, output_img, 100);

            cout<<"SAVED OUTPUT IMAGE FILE: "<<outputImg<<endl<<endl;

            stbi_image_free(rgb_image);
            hipFree(input_img_device);
            hipFree(output_img);
        
}