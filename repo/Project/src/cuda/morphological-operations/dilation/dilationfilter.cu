#include "hip/hip_runtime.h"
#include "dilationfilter.cuh"
#include<cstdio>

__global__ void apply_filter(unsigned char* input, unsigned char* output, int height, int width) {
    extern __shared__ unsigned char shared_memory[];
    unsigned char *image = shared_memory;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int im_x = tx+1;
    int im_y = ty+1;

    int smem_x = (blockDim.x+2);
    int smem_y = (blockDim.y+2);

    int bx = blockIdx.x;
    int by = blockIdx.y;

    int thread_idx = bx*blockDim.x+tx;
    int thread_idy = by*blockDim.y+ty;

    if(tx==0 && ty==0) {
      
        for(int i=1;i<smem_x-1;i++){
          image[NUM_CHANNELS*(i*smem_y+0)] = (by==0) ? 0 : (input[NUM_CHANNELS*((thread_idx+i-1)*height+thread_idy-1)]);
          image[NUM_CHANNELS*(i*smem_y+0)+1] = (by==0) ? 0 : (input[NUM_CHANNELS*((thread_idx+i-1)*height+thread_idy-1)+1]);
          image[NUM_CHANNELS*(i*smem_y+0)+2] = (by==0) ? 0 : (input[NUM_CHANNELS*((thread_idx+i-1)*height+thread_idy-1)+2]); 
        }

        for(int i=1;i<smem_x-1;i++){
          image[NUM_CHANNELS*(i*smem_y+smem_x-1)] = (by==(gridDim.y-1)) ? 0 : (input[NUM_CHANNELS*((thread_idx+i-1)*height+thread_idy+blockDim.y)]);
          image[NUM_CHANNELS*(i*smem_y+smem_x-1)+1] = (by==(gridDim.y-1)) ? 0 :(input[NUM_CHANNELS*((thread_idx+i-1)*height+thread_idy+blockDim.y)+1]);
          image[NUM_CHANNELS*(i*smem_y+smem_x-1)+2] = (by==(gridDim.y-1)) ? 0 : (input[NUM_CHANNELS*((thread_idx+i-1)*height+thread_idy+blockDim.y)+2]); 
        }

        for(int i=1;i<smem_y-1;i++){
          image[NUM_CHANNELS*(0*smem_y+i)] = (bx==0) ? 0 : (input[NUM_CHANNELS*((thread_idx-1)*height+thread_idy+i-1)]);
          image[NUM_CHANNELS*(0*smem_y+i)+1] = (bx==0) ? 0 : (input[NUM_CHANNELS*((thread_idx-1)*height+thread_idy+i-1)+1]);
          image[NUM_CHANNELS*(0*smem_y+i)+2] = (bx==0) ? 0 : (input[NUM_CHANNELS*((thread_idx-1)*height+thread_idy+i-1)+2]); 
        }
    
        for(int i=1;i<smem_y-1;i++){
          image[NUM_CHANNELS*((smem_x-1)*smem_y+i)] = (bx==(gridDim.x-1)) ? 0 : (input[NUM_CHANNELS*((thread_idx+blockDim.x)*height+thread_idy+i-1)]);
          image[NUM_CHANNELS*((smem_x-1)*smem_y+i)+1] = (bx==(gridDim.x-1)) ? 0 : (input[NUM_CHANNELS*((thread_idx+blockDim.x)*height+thread_idy+i-1)+1]);
          image[NUM_CHANNELS*((smem_x-1)*smem_y+i)+2] = (bx==(gridDim.x-1)) ? 0 : (input[NUM_CHANNELS*((thread_idx+blockDim.x)*height+thread_idy+i-1)+2]); 
        }

        //(0,0)
        image[NUM_CHANNELS*((0)*smem_y+0)] = (bx==0 || by==0) ? 0 : (input[NUM_CHANNELS*((thread_idx-1)*height+thread_idy-1)]);
        image[NUM_CHANNELS*((0)*smem_y+0)+1] = (bx==0 || by==0) ? 0 : (input[NUM_CHANNELS*((thread_idx-1)*height+thread_idy-1)+1]);
        image[NUM_CHANNELS*((0)*smem_y+0)+2] = (bx==0 || by==0) ? 0 : (input[NUM_CHANNELS*((thread_idx-1)*height+thread_idy-1)+2]);
        
        //(0,0)
        image[NUM_CHANNELS*((0)*smem_y+smem_y-1)] = (bx==0 || by==gridDim.y-1) ? 0 : (input[NUM_CHANNELS*((thread_idx-1)*height+thread_idy+blockDim.y)]);
        image[NUM_CHANNELS*((0)*smem_y+smem_y-1)+1] = (bx==0 || by==gridDim.y-1) ? 0 : (input[NUM_CHANNELS*((thread_idx-1)*height+thread_idy+blockDim.y)+1]);
        image[NUM_CHANNELS*((0)*smem_y+smem_y-1)+2] = (bx==0 || by==gridDim.y-1) ? 0 : (input[NUM_CHANNELS*((thread_idx-1)*height+thread_idy+blockDim.y)+2]); 

        //(0,0)
        image[NUM_CHANNELS*((smem_x-1)*smem_y+0)] = (bx==gridDim.x-1 || by==0) ? 0 : (input[NUM_CHANNELS*((thread_idx+blockDim.x)*height+thread_idy-1)]);
        image[NUM_CHANNELS*((smem_x-1)*smem_y+0)+1] = (bx==gridDim.x-1 || by==0) ? 0 : (input[NUM_CHANNELS*((thread_idx+blockDim.x)*height+thread_idy-1)+1]);
        image[NUM_CHANNELS*((smem_x-1)*smem_y+0)+2] = (bx==gridDim.x-1 || by==0) ? 0 : (input[NUM_CHANNELS*((thread_idx+blockDim.x)*height+thread_idy-1)+2]); 
    
        //(0,0)
        image[NUM_CHANNELS*((smem_x-1)*smem_y+smem_y-1)] = (bx==gridDim.x-1 || by==gridDim.y-1) ? 0 : (input[NUM_CHANNELS*((thread_idx+blockDim.x)*height+thread_idy+blockDim.y)]);
        image[NUM_CHANNELS*((smem_x-1)*smem_y+smem_y-1)+1] = (bx==gridDim.x-1 || by==gridDim.y-1) ? 0 : (input[NUM_CHANNELS*((thread_idx+blockDim.x)*height+thread_idy+blockDim.y)+1]);
        image[NUM_CHANNELS*((smem_x-1)*smem_y+smem_y-1)+2] = (bx==gridDim.x-1 || by==gridDim.y-1) ? 0 : (input[NUM_CHANNELS*((thread_idx+blockDim.x)*height+thread_idy+blockDim.y)+2]); 

    }

    if(thread_idx<width && thread_idy<height){
        image[NUM_CHANNELS*(im_x*smem_y+im_y)] = (input[NUM_CHANNELS*(thread_idx*height+thread_idy)]);
        image[NUM_CHANNELS*(im_x*smem_y+im_y)+1] = (input[NUM_CHANNELS*(thread_idx*height+thread_idy)+1]);
        image[NUM_CHANNELS*(im_x*smem_y+im_y)+2] = (input[NUM_CHANNELS*(thread_idx*height+thread_idy)+2]);    
    }
    else{
        image[NUM_CHANNELS*(im_x*smem_y+im_y)] = 0;
        image[NUM_CHANNELS*(im_x*smem_y+im_y)+1] = 0;
        image[NUM_CHANNELS*(im_x*smem_y+im_y)+2] = 0;
    }

    __syncthreads();

    unsigned char r_out=image[NUM_CHANNELS*(im_x*smem_y+im_y)];
    unsigned char g_out=image[NUM_CHANNELS*(im_x*smem_y+im_y)+1];
    unsigned char b_out=image[NUM_CHANNELS*(im_x*smem_y+im_y)+2]; 

    r_out = (r_out>image[NUM_CHANNELS*(im_x*smem_y+im_y)])?r_out:image[NUM_CHANNELS*(im_x*smem_y+im_y)];
    r_out = (r_out>image[NUM_CHANNELS*(im_x*smem_y+im_y-1)])?r_out:image[NUM_CHANNELS*(im_x*smem_y+im_y-1)];
    r_out = (r_out>image[NUM_CHANNELS*(im_x*smem_y+im_y+1)])?r_out:image[NUM_CHANNELS*(im_x*smem_y+im_y+1)];
    r_out = (r_out>image[NUM_CHANNELS*((im_x-1)*smem_y+im_y)])?r_out:image[NUM_CHANNELS*((im_x-1)*smem_y+im_y)];
    r_out = (r_out>image[NUM_CHANNELS*((im_x+1)*smem_y+im_y)])?r_out:image[NUM_CHANNELS*((im_x+1)*smem_y+im_y)];
    r_out = (r_out>image[NUM_CHANNELS*((im_x-1)*smem_y+im_y-1)])?r_out:image[NUM_CHANNELS*((im_x-1)*smem_y+im_y-1)];
    r_out = (r_out>image[NUM_CHANNELS*((im_x+1)*smem_y+im_y-1)])?r_out:image[NUM_CHANNELS*((im_x+1)*smem_y+im_y-1)];
    r_out = (r_out>image[NUM_CHANNELS*((im_x-1)*smem_y+im_y+1)])?r_out:image[NUM_CHANNELS*((im_x-1)*smem_y+im_y+1)];
    r_out = (r_out>image[NUM_CHANNELS*((im_x+1)*smem_y+im_y+1)])?r_out:image[NUM_CHANNELS*((im_x+1)*smem_y+im_y+1)];

    g_out = (g_out>image[NUM_CHANNELS*(im_x*smem_y+im_y)+1])?g_out:image[NUM_CHANNELS*(im_x*smem_y+im_y)+1];
    g_out = (g_out>image[NUM_CHANNELS*(im_x*smem_y+im_y-1)+1])?g_out:image[NUM_CHANNELS*(im_x*smem_y+im_y-1)+1];
    g_out = (g_out>image[NUM_CHANNELS*(im_x*smem_y+im_y+1)+1])?g_out:image[NUM_CHANNELS*(im_x*smem_y+im_y+1)+1];
    g_out = (g_out>image[NUM_CHANNELS*((im_x-1)*smem_y+im_y)+1])?g_out:image[NUM_CHANNELS*((im_x-1)*smem_y+im_y)+1];
    g_out = (g_out>image[NUM_CHANNELS*((im_x+1)*smem_y+im_y)+1])?g_out:image[NUM_CHANNELS*((im_x+1)*smem_y+im_y)+1];
    g_out = (g_out>image[NUM_CHANNELS*((im_x-1)*smem_y+im_y-1)+1])?g_out:image[NUM_CHANNELS*((im_x-1)*smem_y+im_y-1)+1];
    g_out = (g_out>image[NUM_CHANNELS*((im_x+1)*smem_y+im_y-1)+1])?g_out:image[NUM_CHANNELS*((im_x+1)*smem_y+im_y-1)+1];
    g_out = (g_out>image[NUM_CHANNELS*((im_x-1)*smem_y+im_y+1)+1])?g_out:image[NUM_CHANNELS*((im_x-1)*smem_y+im_y+1)+1];
    g_out = (g_out>image[NUM_CHANNELS*((im_x+1)*smem_y+im_y+1)+1])?g_out:image[NUM_CHANNELS*((im_x+1)*smem_y+im_y+1)+1];

    b_out = (b_out>image[NUM_CHANNELS*(im_x*smem_y+im_y)+2])?b_out:image[NUM_CHANNELS*(im_x*smem_y+im_y)+2];
    b_out = (b_out>image[NUM_CHANNELS*(im_x*smem_y+im_y-1)+2])?b_out:image[NUM_CHANNELS*(im_x*smem_y+im_y-1)+2];
    b_out = (b_out>image[NUM_CHANNELS*(im_x*smem_y+im_y+1)+2])?b_out:image[NUM_CHANNELS*(im_x*smem_y+im_y+1)+2];
    b_out = (b_out>image[NUM_CHANNELS*((im_x-1)*smem_y+im_y)+2])?b_out:image[NUM_CHANNELS*((im_x-1)*smem_y+im_y)+2];
    b_out = (b_out>image[NUM_CHANNELS*((im_x+1)*smem_y+im_y)+2])?b_out:image[NUM_CHANNELS*((im_x+1)*smem_y+im_y)+2];
    b_out = (b_out>image[NUM_CHANNELS*((im_x-1)*smem_y+im_y-1)+2])?b_out:image[NUM_CHANNELS*((im_x-1)*smem_y+im_y-1)+2];
    b_out = (b_out>image[NUM_CHANNELS*((im_x+1)*smem_y+im_y-1)+2])?b_out:image[NUM_CHANNELS*((im_x+1)*smem_y+im_y-1)+2];
    b_out = (b_out>image[NUM_CHANNELS*((im_x-1)*smem_y+im_y+1)+2])?b_out:image[NUM_CHANNELS*((im_x-1)*smem_y+im_y+1)+2];
    b_out = (b_out>image[NUM_CHANNELS*((im_x+1)*smem_y+im_y+1)+2])?b_out:image[NUM_CHANNELS*((im_x+1)*smem_y+im_y+1)+2];

    output[NUM_CHANNELS*(thread_idx*height+thread_idy)] = r_out;
    output[NUM_CHANNELS*(thread_idx*height+thread_idy)+1] = g_out;
    output[NUM_CHANNELS*(thread_idx*height+thread_idy)+2] = b_out;

}

__host__ void dilationfilter(unsigned char** rgb_image, unsigned char** output_img, int height, int width, int threads_per_block_dim){
   dim3 blockSize(threads_per_block_dim,threads_per_block_dim);
   dim3 gridSize((width+threads_per_block_dim-1)/threads_per_block_dim,(height+threads_per_block_dim-1)/threads_per_block_dim); 

   std::size_t shared_memory = (threads_per_block_dim+2)*(threads_per_block_dim+2)*sizeof(unsigned char)*NUM_CHANNELS;
   hipEvent_t start;
   hipEvent_t stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);

   hipEventRecord(start); //START EVENT

   // Launch the kernel on the device
   apply_filter<<<gridSize,blockSize,shared_memory>>>(*(rgb_image),*(output_img),height,width);

   hipEventRecord(stop); //STOP EVENT
   hipEventSynchronize(stop);

   //Calculate total runtime using events
   float ms;
   hipEventElapsedTime(&ms, start, stop);

   printf("Runtime: %f\n", ms); 
   hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));

    }
    hipDeviceSynchronize();
}