
#include <hip/hip_runtime.h>
__global__ void vscale(const float *a, float *b, unsigned int n) {
    unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;

    //Do this for only n values in the vectors
    if(index<n){
        //compute the eltwise mult here
        b[index] = b[index]*a[index];
    }
}
