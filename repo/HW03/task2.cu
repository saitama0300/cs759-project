
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <ctime>

#define ARRAY_SIZE 16
#define MAX_RAND 100

__global__ void task2(int a, int *output) {
    
    //Calculate the index using block and thread indexes
    int index = blockIdx.x*blockDim.x + threadIdx.x;
  

    //Calculate ax+b for the output
    int x,y;
    x = threadIdx.x;
    y = blockIdx.x;
    output[index] = x*a + y;
    
}

int main() {

    int hA[ARRAY_SIZE];

    std::srand((unsigned int)std::time(NULL));
 
    // Declare pointers that will point to the memory allocated on the device.
    int* dA;
 
    // Allocate memory on the device
    hipMalloc(&dA, sizeof(int)*ARRAY_SIZE);
    
    // Launch the kernel on the device
    int a = std::rand()%MAX_RAND;
    task2<<<2, 8>>>(a, dA);

    // Copy the output array back from the device to the host and print its values
    hipMemcpy(hA, dA, sizeof(int)*ARRAY_SIZE, hipMemcpyDeviceToHost);
    for(int i = 0; i < ARRAY_SIZE-1; i++)
    {
        std::printf("%d ", hA[i]);
    }
    std::printf("%d\n",hA[ARRAY_SIZE-1]);
 
    // Free resources
    hipFree(dA);

    return 0;
}
