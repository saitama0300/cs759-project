#include <cstdio>
#include <cstdlib>
#include <ctime>
#include "matmul.cuh"

#define MAX_RAND 1000

int main(int argc, char* argv[]) {

    std::srand((unsigned int)std::time(NULL));

    std::size_t n = std::atoll(argv[1]);
    std::size_t block_dim = std::atoll(argv[2]); 

    //Allocating memory to host arrays
    int * A = (int *) std::malloc(n*n*sizeof(int));
    int * B = (int *) std::malloc(n*n*sizeof(int));

    int * C = (int *) std::malloc(n*n*sizeof(int)); 
 
    //Populating values within the host arrays
    for(std::size_t i=0;i<n*n;i++) {
        A[i] = (std::rand()%MAX_RAND); //RANDOM VALUE
        B[i] = (std::rand()%MAX_RAND); //RANDOM VALUE
    }  

    // Declare pointers that will point to the memory allocated on the device.
    int *dA, *dB, *dC;
 
    // Allocate memory on the device
    hipMalloc(&dA, sizeof(int)*n*n);
    hipMalloc(&dB, sizeof(int)*n*n);

    //Final result array on device
    hipMalloc(&dC, sizeof(int)*n*n);

    //Copy data from host to device
    hipMemcpy(dA, A, sizeof(int)*n*n, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeof(int)*n*n, hipMemcpyHostToDevice);

    //For recording time:
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start); //START EVENT

    // Launch the kernel on the device
    matmul_1(dA,dB,dC,n,block_dim);

    hipEventRecord(stop); //STOP EVENT
    hipEventSynchronize(stop);

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    // Copy the output array back from the device to the host and print its values
    hipMemcpy(C, dC, sizeof(int)*n*n, hipMemcpyDeviceToHost);

    //Print results
    //for(int i=0;i<n;i++){
      //  for(int j=0;j<n;j++){
        //   std::printf("%d\t", C[i*n+j]);
      //  }
       // printf("\n");
    //}
    std::printf("%d\n", C[0]);
    std::printf("%d\n", C[n*n-1]);
    std::printf("%f\n", ms); 
 
    // Free resources
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    std::free(A);
    std::free(B);
    std::free(C);
    

    //Allocating memory to host arrays
    float * Af = (float *) std::malloc(n*n*sizeof(float));
    float * Bf = (float *) std::malloc(n*n*sizeof(float));

    float * Cf = (float *) std::malloc(n*n*sizeof(float)); 
 
    //Populating values within the host arrays
    for(std::size_t i=0;i<n*n;i++) {
        Af[i] = ((float(std::rand()%MAX_RAND)*2.0)/MAX_RAND)-1.0; //Value between -1.0 and 1.0
        Bf[i] = ((float(std::rand()%MAX_RAND)*2.0)/MAX_RAND)-1.0; //Value between -1.0 and 1.0
    }  

    // Declare pointers that will point to the memory allocated on the device.
    float *dAf, *dBf, *dCf;
 
    // Allocate memory on the device
    hipMalloc(&dAf, sizeof(float)*n*n);
    hipMalloc(&dBf, sizeof(float)*n*n);

    //Final result array on device
    hipMalloc(&dCf, sizeof(float)*n*n);

    //Copy data from host to device
    hipMemcpy(dAf, Af, sizeof(float)*n*n, hipMemcpyHostToDevice);
    hipMemcpy(dBf, Bf, sizeof(float)*n*n, hipMemcpyHostToDevice);

    //For recording time:
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start); //START EVENT

    // Launch the kernel on the device
    matmul_2(dAf,dBf,dCf,n,block_dim);

    hipEventRecord(stop); //STOP EVENT
    hipEventSynchronize(stop);

    // Get the elapsed time in milliseconds
    hipEventElapsedTime(&ms, start, stop);

    // Copy the output array back from the device to the host and print its values
    hipMemcpy(Cf, dCf, sizeof(float)*n*n, hipMemcpyDeviceToHost);

    //Print results
    std::printf("%f\n", Cf[0]);
    std::printf("%f\n", Cf[n*n-1]);
    std::printf("%f\n", ms); 
 
    // Free resources
    hipFree(dAf);
    hipFree(dBf);
    hipFree(dCf);
    std::free(Af);
    std::free(Bf);
    std::free(Cf);

   //Allocating memory to host arrays
   double * Ad = (double *) std::malloc(n*n*sizeof(double));
   double * Bd = (double *) std::malloc(n*n*sizeof(double));

   double * Cd = (double *) std::malloc(n*n*sizeof(double)); 

   //Populating values within the host arrays
   for(std::size_t i=0;i<n*n;i++) {
       Ad[i] = ((double(std::rand()%MAX_RAND)*2.0)/MAX_RAND)-1.0; //Value between -1.0 and 1.0
       Bd[i] = ((double(std::rand()%MAX_RAND)*2.0)/MAX_RAND)-1.0; //Value between -1.0 and 1.0
   }  

   // Declare pointers that will point to the memory allocated on the device.
   double *dAd, *dBd, *dCd;

   // Allocate memory on the device
   hipMalloc(&dAd, sizeof(double)*n*n);
   hipMalloc(&dBd, sizeof(double)*n*n);

   //Final result array on device
   hipMalloc(&dCd, sizeof(double)*n*n);

   //Copy data from host to device
   hipMemcpy(dAd, Ad, sizeof(double)*n*n, hipMemcpyHostToDevice);
   hipMemcpy(dBd, Bd, sizeof(double)*n*n, hipMemcpyHostToDevice);

   //For recording time:
   hipEventCreate(&start);
   hipEventCreate(&stop);

   hipEventRecord(start); //START EVENT

   // Launch the kernel on the device
   matmul_3(dAd,dBd,dCd,n,block_dim);

   hipEventRecord(stop); //STOP EVENT
   hipEventSynchronize(stop);

   // Get the elapsed time in milliseconds
   hipEventElapsedTime(&ms, start, stop);

   // Copy the output array back from the device to the host and print its values
   hipMemcpy(Cd, dCd, sizeof(double)*n*n, hipMemcpyDeviceToHost);

   //Print results
   std::printf("%lf\n", Cd[0]);
   std::printf("%lf\n", Cd[n*n-1]);
   std::printf("%f\n", ms); 

   // Free resources
   hipFree(dAd);
   hipFree(dBd);
   hipFree(dCd);
   std::free(Ad);
   std::free(Bd);
   std::free(Cd); 

   return 0;
}
