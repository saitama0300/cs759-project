
#include <hip/hip_runtime.h>
#include<cstdlib>

template <class TYPE>
__global__ void matmul_kernel(const TYPE* A, const TYPE* B, TYPE* C, unsigned int n) {
        //Reference to Shared Memory
        extern __shared__ char shared_memory_char[];
        TYPE* shared_memory = reinterpret_cast<TYPE*>(shared_memory_char);

        TYPE *aS = shared_memory;
        TYPE *bS = (TYPE *)&aS[blockDim.x*blockDim.y];

        //Calculate indices for block, thread and loop variables
        unsigned int bx = blockIdx.x;
        unsigned int by = blockIdx.y;
      
        unsigned int tx = threadIdx.x;
        unsigned int ty = threadIdx.y;

        unsigned int astart = n*blockDim.x*by;

        unsigned int aend = astart + n - 1;

        unsigned int astep = blockDim.x;

        unsigned int bstart = blockDim.x*bx;
        
        unsigned int bstep = blockDim.x*n;

        //To store the partial sum
        TYPE csub = 0;

        //Loop to execute tiled matrix matrix multiplication
        for(unsigned int a = astart, b = bstart, b_row=0; a<=aend; a+=astep, b+=bstep, b_row+=blockDim.y) {
            //Row & Column transerval index variables
            unsigned int cA = a + tx - blockDim.y * by * n;
            unsigned int rA = blockDim.y * by + ty;

            //Handling corner cases for the shared memory
            if(rA<n && cA<n){
                aS[blockDim.x*ty+tx] = A[a + n*ty + tx];
            }
            else{
                aS[blockDim.x*ty+tx] = 0;
            }

            //Row & Column transerval index variables
            unsigned int cB = blockDim.x * bx + tx;
            unsigned int rB = b_row+ ty;

            //Handling corner cases for the shared memory
            if(rB<n && cB<n) {
                bS[blockDim.y*ty+tx] = B[b + n*ty + tx]; 
            }
            else {
                bS[blockDim.y*ty+tx] = 0; 
            }
            __syncthreads();

            //Calculate partial dot product for Row of A & column of B
            for(unsigned int k=0;k<blockDim.x;k++) {
                csub += aS[ty*blockDim.x+k]*bS[k*blockDim.x+tx];
            }

            __syncthreads();
        }

        //Calculate index variables for storing output of csub to output matrix
        unsigned int c_index = n*blockDim.y*by + blockDim.x*bx;
        unsigned int rC = blockDim.y*by+ty;
        unsigned int cC = blockDim.x*bx+tx;

        //Corner cases handled so matrix C
        if(rC<n && cC<n){
            C[c_index+n*ty+tx] = csub;
        }
    
}

__host__ void matmul_1(const int *A, const int *B, int *C, unsigned int n,
    unsigned int block_dim){
        //Handling Corner cases
        if(block_dim>n){
            block_dim = n;
        }
        
        //Calculating number of blocks based on block_dim
        std::size_t num_blocks = (n+block_dim-1)/block_dim;

        dim3 dimBlock(block_dim, block_dim);
        dim3 dimGrid(num_blocks, num_blocks);   

        //Launch Kernel
        matmul_kernel<int><<<dimGrid,dimBlock,block_dim*block_dim*2*sizeof(int)>>>(A, B, C, n);
        hipDeviceSynchronize();
}
__host__ void matmul_2(const float *A, const float *B, float *C, unsigned int n,
    unsigned int block_dim){
        //Handling Corner cases
        if(block_dim>n){
            block_dim = n;
        }

        //Calculating number of blocks based on block_dim
        std::size_t num_blocks = (n+block_dim-1)/block_dim;

        dim3 dimBlock(block_dim, block_dim);
        dim3 dimGrid(num_blocks, num_blocks);

        //Launch Kernel
        matmul_kernel<float><<<dimGrid,dimBlock,block_dim*block_dim*2*sizeof(float)>>>(A, B, C, n);
        hipDeviceSynchronize();
}
__host__ void matmul_3(const double *A, const double *B, double *C,
    unsigned int n, unsigned int block_dim){
        //Handling corner case
        if(block_dim>n){
            block_dim = n;
        }
        
        //Calculating number of blocks based on block_dim
        std::size_t num_blocks = (n+block_dim-1)/block_dim;

        dim3 dimBlock(block_dim, block_dim);
        dim3 dimGrid(num_blocks, num_blocks);

        //Launch Kernel
        matmul_kernel<double><<<dimGrid,dimBlock,block_dim*block_dim*2*sizeof(double)>>>(A, B, C, n);
        hipDeviceSynchronize();
}