
#include <hip/hip_runtime.h>
__global__ void hillis_steele(float*g_odata, const float*g_idata, float *b_odata, unsigned int n){
    extern volatile __shared__ float t_odata[];

    //Calculate the index to get cumulative thread count
    unsigned int thread_index = blockIdx.x*blockDim.x + threadIdx.x;

    if(thread_index < n) {
        //Bring data to shared memory
        t_odata[threadIdx.x] = g_idata[thread_index];
        int pin=1, pout=0;

        __syncthreads();

        //Only adds up elements within the shared memory (those elements lying between blockIdx.x to blockIdx+1 blocks)
        for(std::size_t i=1;i<blockDim.x;i*=2) {
            //pin and pout toggle their values
            pin = 1 - pin;
            pout = 1 - pout;

            //Add elements
            if(threadIdx.x >= i) {
                t_odata[pout * blockDim.x + threadIdx.x] = t_odata[pin * blockDim.x + threadIdx.x] + t_odata[pin * blockDim.x + threadIdx.x - i];
            }
            else {
                t_odata[pout * blockDim.x + threadIdx.x] = t_odata[pin * blockDim.x + threadIdx.x]; 
            }

            __syncthreads();
        }
        
        //Store temporary data in output array
        g_odata[thread_index] = t_odata[pout*blockDim.x+threadIdx.x];
        __syncthreads();

        //Get cummulative sum across a block (basically sum of all elements in shared memory of this block)
        if(b_odata!=NULL) {
             b_odata[blockIdx.x] = g_odata[blockIdx.x*blockDim.x+blockDim.x-1];
        }
    }

}

__global__ void add_b_odata(float *g_odata, float *b_odata, unsigned int n) {
    //Adds partial block level sums to output array to get final result
    if(blockIdx.x < n-1) {
        g_odata[blockDim.x*(blockIdx.x+1)+threadIdx.x] += b_odata[blockIdx.x];
    }
}

__global__ void add_tail(float *g_odata, float *tail_odata, unsigned int n) {
    //Used to adding tail elements (for case when array input size is not divisible by threads_per_block)
    g_odata[threadIdx.x] += tail_odata[n-1];    
}


__host__ void scan(const float* input, float* output, unsigned int n, unsigned int threads_per_block){
    unsigned int num_tail_data = n % threads_per_block;

    //Case when n is a multiple of threads_per_block
    if(num_tail_data == 0) {
        unsigned int num_blocks = (n+threads_per_block-1)/threads_per_block;
        
        float *b_odata, *b_odata_post_scan;

        //Allocate temporary memory instances to store partial sums across blocks
        hipMalloc((void**)&b_odata, num_blocks * sizeof(float));
        hipMalloc((void**)&b_odata_post_scan, num_blocks * sizeof(float));

        if(n>threads_per_block) {
            //Gives partial sums across all elements in al block
            hillis_steele<<<num_blocks, threads_per_block, 2*threads_per_block*sizeof(float)>>>(output,input,b_odata,n);

            //Apply scan over previous stage partial sum to get Cummulative sum across block sums
            hillis_steele<<<1, num_blocks, 2*num_blocks*sizeof(float)>>>(b_odata_post_scan,b_odata,NULL,num_blocks); 

            //Add the final block sums to all partial outputs of output array to get final result 
            add_b_odata<<<num_blocks, threads_per_block>>> (output,b_odata_post_scan,num_blocks);
        }
        else {
            //Case when number of blocks is only 1
            hillis_steele<<<num_blocks, n, 2*n*sizeof(float)>>>(output,input,b_odata,n); 
        }
        //Free allocated memory
        hipFree(b_odata);
        hipFree(b_odata_post_scan);
    }
    else {

        //n_adjusted is the number of elements not considering the remainder when n is not a multiple of threads_per_block
        unsigned int n_adjusted = n - num_tail_data;
        unsigned num_blocks = (n_adjusted+threads_per_block-1)/threads_per_block;

        float *b_odata, *b_odata_post_scan;

        //Allocate temporary memory instances to store partial sums across blocks
        hipMalloc((void**)&b_odata, num_blocks * sizeof(float));
        hipMalloc((void**)&b_odata_post_scan, num_blocks * sizeof(float));

        //Gives partial sums across all elements in al block
        hillis_steele<<<num_blocks, threads_per_block, 2*threads_per_block*sizeof(float)>>>(output,input,b_odata,n_adjusted);

        //Apply scan over previous stage partial sum to get Cummulative sum across block sums
        hillis_steele<<<1, num_blocks, 2*num_blocks*sizeof(float)>>>(b_odata_post_scan,b_odata,NULL,num_blocks); 

        //Add the final block sums to all partial outputs of output array to get final result
        add_b_odata<<<num_blocks, threads_per_block>>> (output,b_odata_post_scan,num_blocks);

        //Reference the tail elements (the remainder left out if n is not a multiple of threads_per_block)
        const float *tail_idata = &(input[n_adjusted]); //PART OF INPUT DATA
        float *tail_odata = &(output[n_adjusted]); //PART OF OUTPUT DATA

        //Scan the tail ements
        hillis_steele<<<1, num_tail_data, 2*num_tail_data*sizeof(float)>>>(tail_odata, tail_idata,NULL,num_tail_data);

        //Add scanned cummulative block sums to tail elements to have consistent result
        add_tail<<<1, num_tail_data>>>(tail_odata, b_odata_post_scan, num_blocks);

        //Free memory allocated
        hipFree(b_odata);
        hipFree(b_odata_post_scan);
    }

    //Call to device synchronize
    hipDeviceSynchronize();
}
