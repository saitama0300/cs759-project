
#include <hip/hip_runtime.h>
__global__ void matmul_kernel(const float* A, const float* B, float* C, size_t n) {

    //Fetch the index from thread and block IDs
    std::size_t thread_index = threadIdx.x + blockIdx.x*blockDim.x;
    std::size_t i = (thread_index)/n; 
    std::size_t j = (thread_index)%n;

    float sum = 0;

    if(i<n && j<n) {
        //Compute Sum across index k for i and j: i->Row; j-> Column

        for(std::size_t k=0;k<n;k++){
            sum += A[i*n+k]*B[k*n+j];
        }

        //Send the sum back to output in device memory
        C[i*n+j] = sum;
    }
}

void matmul(const float* A, const float* B, float* C, size_t n, unsigned int threads_per_block){

    //Calculate the number of blocks for 1D kernel configuration
    std::size_t num_blocks = (threads_per_block-1+(n*n))/threads_per_block;

    // Launch the kernel on the device
    matmul_kernel<<<num_blocks,threads_per_block>>> (A, B, C, n);

    hipDeviceSynchronize();
}
