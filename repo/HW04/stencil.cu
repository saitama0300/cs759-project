
#include <hip/hip_runtime.h>

__global__ void stencil_kernel(const float* image, const float* mask, float* output, unsigned int n, unsigned int R)
{
    //Image, mask & output shared memory instances
    extern __shared__ float shared_memory[];

    float *image_shared = shared_memory;
    float *mask_shared = (float *)&image_shared[2*R+blockDim.x];
    float *output_shared = (float *)&mask_shared[2*R+1];
    //Calculate thread index
    long long int thread_index = threadIdx.x + blockIdx.x*blockDim.x;

    if(threadIdx.x==0) {
            if(thread_index-R<0){
                for(long long int i=0;i<R;i++){
                    image_shared[i] = 1.0; 
                }
            }
            else{
                for(long long int i=0;i<R;i++){
                    image_shared[i] = image[thread_index-R+i];
                }
            }
            image_shared[R] = image[thread_index];
    }
    else if(threadIdx.x==blockDim.x-1) {
        
            if(thread_index+R>=n){
                for(long long int i=0;i<R;i++) {
                    image_shared[R+blockDim.x+i] = 1.0;
                }
            }
            else {
                for(long long int i=0;i<R;i++) {
                    image_shared[blockDim.x+R+i] = image[thread_index+1+i];
                }
            }
            image_shared[blockDim.x+R-1] = image[thread_index]; 
    }
    else{
        image_shared[R+threadIdx.x] = image[thread_index];
    }

    //CORNER CASE HANDLED - for n not being a multiple of number of threads per block
    if(thread_index==n-1) {
        for(long long int i=0;i<R;i++) {
            image_shared[R+threadIdx.x+1+i] = 1.0;
        }
    }

    output_shared[threadIdx.x] = 0;

    if(threadIdx.x<=2*R) {
        mask_shared[threadIdx.x] = mask[threadIdx.x];
    }

    //Wait for all threads to finish copying 
    __syncthreads(); 

    if(thread_index<n) {
        //Compute the output of stencil
        for(int j= 0;j<=2*R;j++) {
                output_shared[threadIdx.x] += image_shared[threadIdx.x+j]*mask_shared[j];     
        }

        //Send result of output shared memory to output in global device memory
        __syncthreads(); 
        
        output[thread_index] = output_shared[threadIdx.x];  
       
    }
}

__host__ void stencil(const float* image, const float* mask, float* output, unsigned int n, unsigned int R, unsigned int threads_per_block)
{
    //Calculate the number of blocks
    std::size_t num_blocks = (threads_per_block-1+n)/threads_per_block;

    //Call kernel with dynamic shared memory
    stencil_kernel<<<num_blocks,threads_per_block,((threads_per_block+2*R)*2+1)*sizeof(float)>>> (image, mask, output, n, R);

    //Cuda device synchronizer
    hipDeviceSynchronize();
}