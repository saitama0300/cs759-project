#include "hip/hip_runtime.h"
#include "medianfilter.cuh"
#include<cstdio>

__global__ void apply_filter(unsigned char* input, unsigned char* output, int height, int width) {
    extern __shared__ unsigned char shared_memory[];
    unsigned char *image = shared_memory;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int im_x = tx+1;
    int im_y = ty+1;

    int smem_x = (blockDim.x+2);
    int smem_y = (blockDim.y+2);

    int bx = blockIdx.x;
    int by = blockIdx.y;

    int thread_idx = bx*blockDim.x+tx;
    int thread_idy = by*blockDim.y+ty;

    if(tx==0 && ty==0) {
      
        for(int i=1;i<smem_x-1;i++){
          image[NUM_CHANNELS*(i*smem_y+0)] = (by==0) ? 0 : (input[NUM_CHANNELS*((thread_idx+i-1)*height+thread_idy-1)]);
          image[NUM_CHANNELS*(i*smem_y+0)+1] = (by==0) ? 0 : (input[NUM_CHANNELS*((thread_idx+i-1)*height+thread_idy-1)+1]);
          image[NUM_CHANNELS*(i*smem_y+0)+2] = (by==0) ? 0 : (input[NUM_CHANNELS*((thread_idx+i-1)*height+thread_idy-1)+2]);
        }

        for(int i=1;i<smem_x-1;i++){
          image[NUM_CHANNELS*(i*smem_y+smem_x-1)] = (by==(gridDim.y-1)) ? 0 : (input[NUM_CHANNELS*((thread_idx+i-1)*height+thread_idy+blockDim.y)]);
          image[NUM_CHANNELS*(i*smem_y+smem_x-1)+1] = (by==(gridDim.y-1)) ? 0 :(input[NUM_CHANNELS*((thread_idx+i-1)*height+thread_idy+blockDim.y)+1]);
          image[NUM_CHANNELS*(i*smem_y+smem_x-1)+2] = (by==(gridDim.y-1)) ? 0 : (input[NUM_CHANNELS*((thread_idx+i-1)*height+thread_idy+blockDim.y)+2]); 
        }

        for(int i=1;i<smem_y-1;i++){
          image[NUM_CHANNELS*(0*smem_y+i)] = (bx==0) ? 0 : (input[NUM_CHANNELS*((thread_idx-1)*height+thread_idy+i-1)]);
          image[NUM_CHANNELS*(0*smem_y+i)+1] = (bx==0) ? 0 : (input[NUM_CHANNELS*((thread_idx-1)*height+thread_idy+i-1)+1]);
          image[NUM_CHANNELS*(0*smem_y+i)+2] = (bx==0) ? 0 : (input[NUM_CHANNELS*((thread_idx-1)*height+thread_idy+i-1)+2]);
        }
    
        for(int i=1;i<smem_y-1;i++){
          image[NUM_CHANNELS*((smem_x-1)*smem_y+i)] = (bx==(gridDim.x-1)) ? 0 : (input[NUM_CHANNELS*((thread_idx+blockDim.x)*height+thread_idy+i-1)]);
          image[NUM_CHANNELS*((smem_x-1)*smem_y+i)+1] = (bx==(gridDim.x-1)) ? 0 : (input[NUM_CHANNELS*((thread_idx+blockDim.x)*height+thread_idy+i-1)+1]);
          image[NUM_CHANNELS*((smem_x-1)*smem_y+i)+2] = (bx==(gridDim.x-1)) ? 0 : (input[NUM_CHANNELS*((thread_idx+blockDim.x)*height+thread_idy+i-1)+2]); 
        }

        //(0,0)
        image[NUM_CHANNELS*((0)*smem_y+0)] = (bx==0 || by==0) ? 0 : (input[NUM_CHANNELS*((thread_idx-1)*height+thread_idy-1)]);
        image[NUM_CHANNELS*((0)*smem_y+0)+1] = (bx==0 || by==0) ? 0 : (input[NUM_CHANNELS*((thread_idx-1)*height+thread_idy-1)+1]);
        image[NUM_CHANNELS*((0)*smem_y+0)+2] = (bx==0 || by==0) ? 0 : (input[NUM_CHANNELS*((thread_idx-1)*height+thread_idy-1)+2]);
        
        //(0,0)
        image[NUM_CHANNELS*((0)*smem_y+smem_y-1)] = (bx==0 || by==gridDim.y-1) ? 0 : (input[NUM_CHANNELS*((thread_idx-1)*height+thread_idy+blockDim.y)]);
        image[NUM_CHANNELS*((0)*smem_y+smem_y-1)+1] = (bx==0 || by==gridDim.y-1) ? 0 : (input[NUM_CHANNELS*((thread_idx-1)*height+thread_idy+blockDim.y)+1]);
        image[NUM_CHANNELS*((0)*smem_y+smem_y-1)+2] = (bx==0 || by==gridDim.y-1) ? 0 : (input[NUM_CHANNELS*((thread_idx-1)*height+thread_idy+blockDim.y)+2]); 

        //(0,0)
        image[NUM_CHANNELS*((smem_x-1)*smem_y+0)] = (bx==gridDim.x-1 || by==0) ? 0 : (input[NUM_CHANNELS*((thread_idx+blockDim.x)*height+thread_idy-1)]);
        image[NUM_CHANNELS*((smem_x-1)*smem_y+0)+1] = (bx==gridDim.x-1 || by==0) ? 0 : (input[NUM_CHANNELS*((thread_idx+blockDim.x)*height+thread_idy-1)+1]);
        image[NUM_CHANNELS*((smem_x-1)*smem_y+0)+2] = (bx==gridDim.x-1 || by==0) ? 0 : (input[NUM_CHANNELS*((thread_idx+blockDim.x)*height+thread_idy-1)+2]); 
    
        //(0,0)
        image[NUM_CHANNELS*((smem_x-1)*smem_y+smem_y-1)] = (bx==gridDim.x-1 || by==gridDim.y-1) ? 0 : (input[NUM_CHANNELS*((thread_idx+blockDim.x)*height+thread_idy+blockDim.y)]);
        image[NUM_CHANNELS*((smem_x-1)*smem_y+smem_y-1)+1] = (bx==gridDim.x-1 || by==gridDim.y-1) ? 0 : (input[NUM_CHANNELS*((thread_idx+blockDim.x)*height+thread_idy+blockDim.y)+1]);
        image[NUM_CHANNELS*((smem_x-1)*smem_y+smem_y-1)+2] = (bx==gridDim.x-1 || by==gridDim.y-1) ? 0 : (input[NUM_CHANNELS*((thread_idx+blockDim.x)*height+thread_idy+blockDim.y)+2]); 

    }

    if(thread_idx<width && thread_idy<height){
        image[NUM_CHANNELS*(im_x*smem_y+im_y)] = (input[NUM_CHANNELS*(thread_idx*height+thread_idy)]);
        image[NUM_CHANNELS*(im_x*smem_y+im_y)+1] = (input[NUM_CHANNELS*(thread_idx*height+thread_idy)+1]);
        image[NUM_CHANNELS*(im_x*smem_y+im_y)+2] = (input[NUM_CHANNELS*(thread_idx*height+thread_idy)+2]);    
    }
    else{
        image[NUM_CHANNELS*(im_x*smem_y+im_y)] = 0;
        image[NUM_CHANNELS*(im_x*smem_y+im_y)+1] = 0;
        image[NUM_CHANNELS*(im_x*smem_y+im_y)+2] = 0;
    }

    __syncthreads();
    unsigned char elements_r[NUM_ELEMENTS] =  {image[NUM_CHANNELS*((im_x-1)*smem_y+im_y-1)],
        image[NUM_CHANNELS*(im_x*smem_y+im_y-1)],
        image[NUM_CHANNELS*((im_x+1)*smem_y+im_y-1)],
        image[NUM_CHANNELS*((im_x-1)*smem_y+im_y)],
        image[NUM_CHANNELS*((im_x)*smem_y+im_y)],
        image[NUM_CHANNELS*((im_x+1)*smem_y+im_y+1)],
        image[NUM_CHANNELS*((im_x-1)*smem_y+im_y+1)],
        image[NUM_CHANNELS*((im_x)*smem_y+im_y+1)],
        image[NUM_CHANNELS*((im_x+1)*smem_y+im_y+1)]
    };
    unsigned char elements_g[NUM_ELEMENTS] =  {image[NUM_CHANNELS*((im_x-1)*smem_y+im_y-1)+1],
        image[NUM_CHANNELS*(im_x*smem_y+im_y-1)+1],
        image[NUM_CHANNELS*((im_x+1)*smem_y+im_y-1)+1],
        image[NUM_CHANNELS*((im_x-1)*smem_y+im_y)+1],
        image[NUM_CHANNELS*((im_x)*smem_y+im_y)+1],
        image[NUM_CHANNELS*((im_x+1)*smem_y+im_y+1)+1],
        image[NUM_CHANNELS*((im_x-1)*smem_y+im_y+1)+1],
        image[NUM_CHANNELS*((im_x)*smem_y+im_y+1)+1],
        image[NUM_CHANNELS*((im_x+1)*smem_y+im_y+1)+1]
    };
    unsigned char elements_b[NUM_ELEMENTS] =  {image[NUM_CHANNELS*((im_x-1)*smem_y+im_y-1)+2],
        image[NUM_CHANNELS*(im_x*smem_y+im_y-1)+2],
        image[NUM_CHANNELS*((im_x+1)*smem_y+im_y-1)+2],
        image[NUM_CHANNELS*((im_x-1)*smem_y+im_y)+2],
        image[NUM_CHANNELS*((im_x)*smem_y+im_y)+2],
        image[NUM_CHANNELS*((im_x+1)*smem_y+im_y+1)+2],
        image[NUM_CHANNELS*((im_x-1)*smem_y+im_y+1)+2],
        image[NUM_CHANNELS*((im_x)*smem_y+im_y+1)+2],
        image[NUM_CHANNELS*((im_x+1)*smem_y+im_y+1)+2]
    };
     
    for(int i=0;i<=NUM_ELEMENTS/2;i++){
        for(int j=i+1;j<NUM_ELEMENTS;j++){
            unsigned char temp;
            if(elements_r[j]<elements_r[i]){
                temp = elements_r[i];
                elements_r[i] = elements_r[j];
                elements_r[j] = temp;
            }
            if(elements_g[j]<elements_g[i]){
                temp = elements_g[i];
                elements_g[i] = elements_g[j];
                elements_g[j] = temp;
            }
            if(elements_b[j]<elements_b[i]){
                temp = elements_b[i];
                elements_b[i] = elements_b[j];
                elements_b[j] = temp;
            }
        }
    }

    output[NUM_CHANNELS*(thread_idx*height+thread_idy)] = elements_r[NUM_ELEMENTS/2];
    output[NUM_CHANNELS*(thread_idx*height+thread_idy)+1] = elements_g[NUM_ELEMENTS/2];
    output[NUM_CHANNELS*(thread_idx*height+thread_idy)+2] = elements_b[NUM_ELEMENTS/2];

}

__host__ void medianfilter(unsigned char** rgb_image, unsigned char** output_img, int height, int width, int threads_per_block_dim){
   dim3 blockSize(threads_per_block_dim,threads_per_block_dim);
   dim3 gridSize((width+threads_per_block_dim-1)/threads_per_block_dim,(height+threads_per_block_dim-1)/threads_per_block_dim); 

   std::size_t shared_memory = (threads_per_block_dim+2)*(threads_per_block_dim+2)*sizeof(unsigned char)*NUM_CHANNELS;
   hipEvent_t start;
   hipEvent_t stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);

   hipEventRecord(start); //START EVENT

   // Launch the kernel on the device
   apply_filter<<<gridSize,blockSize,shared_memory>>>(*(rgb_image),*(output_img),height,width);

   hipEventRecord(stop); //STOP EVENT
   hipEventSynchronize(stop);

   //Calculate total runtime using events
   float ms;
   hipEventElapsedTime(&ms, start, stop);

   printf("Runtime: %f\n", ms); 
   hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));

    }
    hipDeviceSynchronize();
}