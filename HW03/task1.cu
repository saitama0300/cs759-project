
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

//Kernel defined here
__global__ void factorial() {
   
    int fact = 1;
    //Calculating Factorial
    for(int i=threadIdx.x+1;i>1;i--){
        fact = fact*i;
    }   

    //Printing Result
    std::printf("%d!=%d\n",threadIdx.x+1,fact);
}

int main() {
    
    //Calling Kernel
    factorial<<<1,8>>> ();

    //Calling Device Sync function
    hipDeviceSynchronize();

    return 0;
}
