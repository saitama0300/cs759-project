#include <cstdio>
#include <cstdlib>
#include <ctime>
#include "vscale.cuh"

#define NUM_THREADS_PER_BLOCK 512
#define MAX_RAND 1000

int main(int argc, char* argv[]) {

    std::srand((unsigned int)std::time(NULL));
    
    std::size_t n = std::atoll(argv[1]);

    //Allocating memory to host arrays
    float * a = (float *) std::malloc(n*sizeof(float));
    float * b = (float *) std::malloc(n*sizeof(float));
 
    //Populating values within the host arrays
    for(std::size_t i=0;i<n;i++) {
        a[i] = ((float(std::rand()%MAX_RAND)*20.0)/MAX_RAND)-10.0; //Value between -10.0 and 10.0
        b[i] = ((float(std::rand()%MAX_RAND)*1.0)/MAX_RAND); //Value between 0.0 & 1.0
    }  

    // Declare pointers that will point to the memory allocated on the device.
    float *dA, *dB;
 
    // Allocate memory on the device
    hipMalloc(&dA, sizeof(float)*n);
    hipMalloc(&dB, sizeof(float)*n);

    //Copy data from host to device
    hipMemcpy(dA, a, sizeof(float)*n, hipMemcpyHostToDevice);
    hipMemcpy(dB, b, sizeof(float)*n, hipMemcpyHostToDevice);

    std::size_t num_blocks = ((n+NUM_THREADS_PER_BLOCK-1)/NUM_THREADS_PER_BLOCK); 

    //For recording time:
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start); //START EVENT

    // Launch the kernel on the device
    vscale<<<num_blocks, NUM_THREADS_PER_BLOCK>>>(dA,dB,n);

    hipEventRecord(stop); //STOP EVENT
    hipEventSynchronize(stop);

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    // Copy the output array back from the device to the host and print its values
    hipMemcpy(b, dB, sizeof(float)*n, hipMemcpyDeviceToHost);

    //Print results
    std::printf("%f\n", ms); 
    std::printf("%f\n", b[0]);
    std::printf("%f\n", b[n-1]);
 
    // Free resources
    hipFree(dA);
    hipFree(dB);
    std::free(a);
    std::free(b);

    return 0;
}
