#include <cstdio>
#include <cstdlib>
#include <ctime>
#include "matmul.cuh"

#define MAX_RAND 1000

int main(int argc, char* argv[]) {

    std::srand((unsigned int)std::time(NULL));
    
    std::size_t n = std::atoll(argv[1]);
    std::size_t threads_per_block = std::atoll(argv[2]); 

    //Allocating memory to host arrays
    float * A = (float *) std::malloc(n*n*sizeof(float));
    float * B = (float *) std::malloc(n*n*sizeof(float));

    float * C = (float *) std::malloc(n*n*sizeof(float)); 
 
    //Populating values within the host arrays
    for(std::size_t i=0;i<n*n;i++) {
        A[i] = ((float(std::rand()%MAX_RAND)*2.0)/MAX_RAND)-1.0; //Value between -1.0 and 1.0
        B[i] = ((float(std::rand()%MAX_RAND)*2.0)/MAX_RAND)-1.0; //Value between -1.0 and 1.0
    }  

    // Declare pointers that will point to the memory allocated on the device.
    float *dA, *dB, *dC;
 
    // Allocate memory on the device
    hipMalloc(&dA, sizeof(float)*n*n);
    hipMalloc(&dB, sizeof(float)*n*n);

    //Final result array on device
    hipMalloc(&dC, sizeof(float)*n*n);

    //Copy data from host to device
    hipMemcpy(dA, A, sizeof(float)*n*n, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeof(float)*n*n, hipMemcpyHostToDevice);

    //For recording time:
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start); //START EVENT

    // Launch the kernel on the device
    matmul(dA,dB,dC,n,threads_per_block);

    hipEventRecord(stop); //STOP EVENT
    hipEventSynchronize(stop);

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    // Copy the output array back from the device to the host and print its values
    hipMemcpy(C, dC, sizeof(float)*n*n, hipMemcpyDeviceToHost);

    //Print results
    std::printf("%f\n", C[n*n-1]);
    std::printf("%f\n", ms); 
 
    // Free resources
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    std::free(A);
    std::free(B);
    std::free(C);

    return 0;
}
