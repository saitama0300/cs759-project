#include <cstdio>
#include <cstdlib>
#include <ctime>
#include "stencil.cuh"

#define MAX_RAND 1000

int main(int argc, char* argv[]) {

    std::srand((unsigned int)std::time(NULL));
    
    std::size_t n = std::atoll(argv[1]);
    std::size_t R = std::atoll(argv[2]); 
    std::size_t threads_per_block = std::atoll(argv[3]); 

    //Allocating memory to host arrays
    float * image = (float *) std::malloc(n*sizeof(float));
    float * mask = (float *) std::malloc((2*R+1)*sizeof(float));

    float * output = (float *) std::malloc(n*sizeof(float)); 
 
    //Populating values within the host arrays
    for(std::size_t i=0;i<n;i++) {
        image[i] = ((float(std::rand()%MAX_RAND)*2.0)/MAX_RAND)-1.0; //Value between -1.0 and 1.0
    }  

    for(std::size_t i=0;i<=2*R;i++) {
        mask[i] = ((float(std::rand()%MAX_RAND)*2.0)/MAX_RAND)-1.0; //Value between -1.0 and 1.0
    } 

    // Declare pointers that will point to the memory allocated on the device.
    float *dImage, *dMask, *dOutput;
 
    // Allocate memory on the device
    hipMalloc(&dImage, sizeof(float)*n);
    hipMalloc(&dMask, sizeof(float)*(2*R+1));

    //Final result array on device
    hipMalloc(&dOutput, sizeof(float)*n);

    //Copy data from host to device
    hipMemcpy(dImage, image, sizeof(float)*n, hipMemcpyHostToDevice);
    hipMemcpy(dMask, mask, sizeof(float)*(2*R+1), hipMemcpyHostToDevice);

    //For recording time:
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start); //START EVENT

    // Launch the kernel on the device
    stencil(dImage,dMask,dOutput,n,R,threads_per_block);

    hipEventRecord(stop); //STOP EVENT
    hipEventSynchronize(stop);

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    // Copy the output array back from the device to the host and print its values
    hipMemcpy(output, dOutput, sizeof(float)*n, hipMemcpyDeviceToHost);

    //Print results
    std::printf("%f\n", output[n-1]);
    std::printf("%f\n", ms); 
 
    // Free resources
    hipFree(dImage);
    hipFree(dMask);
    hipFree(dOutput);
    std::free(image);
    std::free(mask);
    std::free(output);

    return 0;
}
