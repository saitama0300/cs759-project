#include <cstdio>
#include <cstdlib>
#include <ctime>
#include "scan.cuh"

#define MAX_RAND 1000

int main(int argc, char* argv[]) {
 
    std::srand((unsigned int)std::time(NULL));

    //Command line options
    std::size_t n = std::atoll(argv[1]);
    std::size_t threads_per_block = std::atoll(argv[2]); 

    // Declare pointers that will point to the memory allocated 
    float *input, *output;
   
    // Allocate memory (Managed)
    hipMallocManaged(&input, sizeof(float)*n);
    hipMallocManaged(&output, sizeof(float)*n);

    //Populating values within the host arrays
    for(std::size_t i=0;i<n;i++) {
        input[i] = ((float(std::rand()%MAX_RAND)*2.0)/MAX_RAND)-1.0; //RANDOM VALUE
    }  
    
    //For recording time:

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start); //START EVENT

    // Call the scan funtion from host
    scan(input, output, n, threads_per_block);

    hipEventRecord(stop); //STOP EVENT
    hipEventSynchronize(stop);

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    std::printf("%f\n", output[n-1]); 
    std::printf("%f\n", ms); 
 
    // Free resources
    hipFree(input);
    hipFree(output);
}