
#include <hip/hip_runtime.h>
#include <hipblas.h>

void mmul(hipblasHandle_t handle, const float* A, const float* B, float* C, int n){
    float alpha = 1.0;
    float beta = 1.0;
    //Callmto cublas function for GEMM
    hipblasSgemm(handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            n, n, n,
            &alpha, // We have to add AB to final result, so we use Beta as 1
            A, n,
            B, n,
            &beta, // We have to add C, so we use Beta as 1
            C, n);

    //Device Synchronize Call
    hipDeviceSynchronize();
}
