#include <cstdio>
#include <cstdlib>
#include <ctime>
#include "mmul.h"

#define MAX_RAND 1000

int main(int argc, char* argv[]) {

    std::srand((unsigned int)std::time(NULL));

    std::size_t n = std::atoll(argv[1]);
    std::size_t n_tests = std::atoll(argv[2]); 

    // Declare pointers that will point to the memory allocated on the device.
    float *A, *B, *C;

    // Allocate memory (Managed)
    hipMallocManaged(&A, sizeof(float)*n*n);
    hipMallocManaged(&B, sizeof(float)*n*n);
    hipMallocManaged(&C, sizeof(float)*n*n);

    //Populating values within the host arrays
    for(std::size_t i=0;i<n*n;i++) {
        A[i] = ((float(std::rand()%MAX_RAND)*2.0)/MAX_RAND)-1.0; //RANDOM VALUE
        B[i] = ((float(std::rand()%MAX_RAND)*2.0)/MAX_RAND)-1.0; //RANDOM VALUE
        C[i] = ((float(std::rand()%MAX_RAND)*2.0)/MAX_RAND)-1.0; //RANDOM VALUE
    }  

    //Cublas handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    //Runtime calculation variables
    float total_runtime=0;
    float average_runtime=0;

    for(std::size_t i=0;i<n_tests;i++){
        //For recording time:

        hipEvent_t start;
        hipEvent_t stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start); //START EVENT

        // Call to the function
        mmul(handle, A, B, C, n);

        hipEventRecord(stop); //STOP EVENT
        hipEventSynchronize(stop);

        // Get the elapsed time in milliseconds
        float ms;
        hipEventElapsedTime(&ms, start, stop);

        total_runtime = total_runtime + ms;
    }

    //Get average runtime and print it
    average_runtime = total_runtime/n_tests;

    std::printf("%f\n", average_runtime); 
 
    // Free resources
    hipblasDestroy(handle);
    hipFree(A);
    hipFree(B);
    hipFree(C);
}